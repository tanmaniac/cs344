/**
 *  Implementation of Hillis-Steele parallel prefix scan.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void exclusiveScanKernel(T *dataOut, const T *dataIn, const size_t dataSize) {
    extern __shared__ T shmData[];

    int thId = threadIdx.x;
    int bufA = 0, bufB = 1;

    // Load everything into shared memory. We need to copy twice to fill the shared memory space
    shmData[bufA * dataSize + thId] = (thId == 0) ? 0 : dataIn[thId - 1];
    shmData[bufB * dataSize + thId] = (thId == 0) ? 0 : dataIn[thId - 1];
    __syncthreads();

    for (int offset = 1; offset < dataSize; offset <<= 1) {
        // Swap which side of the buffer we're writing into
        bufA = 1 - bufA;
        bufB = 1 - bufA;
        // Do scan step
        if (thId >= offset) {
            shmData[bufA * dataSize + thId] = shmData[bufB * dataSize + thId] + 
                                                    shmData[bufB * dataSize + thId - offset];
        } else {
            shmData[bufA * dataSize + thId] = shmData[bufB * dataSize + thId];
        }

        __syncthreads();
    }
    // Write to output array
    dataOut[thId] = shmData[bufA * dataSize + thId];
}

//template <typename T>
//void launchScanKernel(T *h_dataOut, const T *h_dataIn, const size_t dataSize) {
void launchScanKernel(int *h_dataOut, const int *h_dataIn, const size_t dataSize) {
    const size_t dataBytes = dataSize * sizeof(int);

    // Declare GPU memory pointers
    int *d_dataOut, *d_dataIn;

    // Allocate GPU memory
    hipMalloc((void **) &d_dataOut, dataBytes);
    hipMalloc((void **) &d_dataIn, dataBytes);

    hipMemcpy(d_dataIn, h_dataIn, dataBytes, hipMemcpyHostToDevice);

    exclusiveScanKernel<<<1, dataSize, 2 * dataBytes>>>(d_dataOut, d_dataIn, dataSize);

    // Copy back from GPU to CPU
    hipMemcpy(h_dataOut, d_dataOut, dataBytes, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_dataIn);
    hipFree(d_dataOut);
}

//template void launchScanKernel<int>(int *h_dataOut, const int *h_dataIn, const size_t dataSize);